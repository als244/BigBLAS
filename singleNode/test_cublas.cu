
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hipblaslt.h>
#include <hiprand/hiprand.h>
#include <sys/time.h>

struct matrixMeta {
	size_t rows;
	size_t cols;
	int typeID;
};

int main(int argc, char * argv[]){

	if (argc != 4){
		fprintf(stderr, "Wrong number of args\n");
		exit(1);
	}

	char * fileNameA = argv[1];
	char * fileNameB = argv[2];
	char * fileNameC = argv[3];

	char * pathA, *pathAMeta, *pathB, *pathBMeta;

	asprintf(&pathA, "/home/shein/Documents/grad_school/research/BigBLAS/data/%s", fileNameA);
	asprintf(&pathAMeta, "/home/shein/Documents/grad_school/research/BigBLAS/data/%s.metadata", fileNameA);
	asprintf(&pathB, "/home/shein/Documents/grad_school/research/BigBLAS/data/%s", fileNameB);
	asprintf(&pathBMeta, "/home/shein/Documents/grad_school/research/BigBLAS/data/%s.metadata", fileNameB);

	FILE * fpA, *fpAMeta, * fpB, * fpBMeta;

	struct matrixMeta A_metadata, B_metadata;

	// get A metadata
	fpAMeta = fopen(pathAMeta, "r");
	free(pathAMeta);
	fread(&A_metadata, sizeof(struct matrixMeta), 1, fpAMeta);
	fclose(fpAMeta);

	// get B metadata
	fpBMeta = fopen(pathBMeta, "r");
	free(pathBMeta);
	fread(&B_metadata, sizeof(struct matrixMeta), 1, fpBMeta);
	fclose(fpBMeta);

	size_t M = A_metadata.rows;
	size_t K = A_metadata.cols;
	size_t N = B_metadata.cols;

	if (K != B_metadata.rows){
		fprintf(stderr, "Matrix dims do not align!\n");
		exit(1);
	}

	// Create files for output C matrix
	char * pathC, * pathCMeta;
	asprintf(&pathC, "/home/shein/Documents/grad_school/research/BigBLAS/data/output_%s", fileNameC);
	asprintf(&pathCMeta, "/home/shein/Documents/grad_school/research/BigBLAS/data/output_%s.metadata", fileNameC);

	FILE * fpC, *fpCMeta;

	// write metadata struct
	struct matrixMeta C_metadata = {M, N, A_metadata.typeID};
	fpCMeta = fopen(pathCMeta, "w+");
	free(pathCMeta);
	fwrite(&C_metadata, sizeof(struct matrixMeta), 1, fpCMeta);
	fclose(fpCMeta);


	// MATRICES SMALL ENOUGHT TO READ INTO MEMORY!
	float * A, *B;

	A = (float *) calloc(M * K, sizeof(float));
	B = (float *) calloc(K * N, sizeof(float));
	
	// read A
	fpA = fopen(pathA, "r");
	free(pathA);
	fread(A, sizeof(float), M * K, fpA);
	fclose(fpA);

	// read B
	fpB = fopen(pathB, "r");
	free(pathB);
	fread(B, sizeof(float), K * N, fpB);
	fclose(fpB);



	/* SETTING CUBLAS UP */
	void *d_A, *d_B, *d_C;

	hipMalloc(&d_A, M * K * sizeof(float));
	hipMalloc(&d_B, K * M * sizeof(float));
	hipMalloc(&d_C, M * N * sizeof(float));


	// copy over input matrices to device
	hipMemcpy(d_A, A, M * K * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, K * N * sizeof(float), hipMemcpyHostToDevice);

	// free the input matrices
	free(A);
	free(B);

	// allocate output matrix
	float * C = (float *) calloc(M * N, sizeof(float));

	
	// deal with cuBLAS structs
	hipblasStatus_t status;
	hipblasLtHandle_t handle;
	status = hipblasLtCreate(&handle);


	hipblasOperation_t transa = HIPBLAS_OP_T;
	hipblasOperation_t transb = HIPBLAS_OP_N;

	hipblasLtMatrixLayout_t Adesc, Bdesc, Cdesc, Ddesc;

	hipblasLtMatmulDesc_t matmulDesc;

	status = hipblasLtMatmulDescCreate(&matmulDesc, HIPBLAS_COMPUTE_32F_FAST_16F, HIP_R_32F);
	status = hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa));
	status = hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transb));

	// A Transposed (from row-major to column-major), not B/D (but still held in col-major format internally)
	// m and k must be multiples of 4, perferablly multiples of 16
	status = hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_32F, K, M, K);
	status = hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_32F, K, N, K);
	status = hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_32F, M, N, M);
	status = hipblasLtMatrixLayoutCreate(&Ddesc, HIP_R_32F, M, N, M);


	hipblasLtMatmulPreference_t pref;
	status = hipblasLtMatmulPreferenceCreate(&pref);
	// ALLOW workspace mem...
	const size_t workspaceBytes = 0;
	status = hipblasLtMatmulPreferenceSetAttribute(pref, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceBytes, sizeof(workspaceBytes));

	int algoCount = 1;
	int retAlgoCount = 0;

	hipblasLtMatmulHeuristicResult_t heuristicResultsArray = {};

	status = hipblasLtMatmulAlgoGetHeuristic(handle, matmulDesc, Adesc, Bdesc, Cdesc, Ddesc, pref, algoCount, &heuristicResultsArray, &retAlgoCount);

	hipblasLtMatmulAlgo_t algo = heuristicResultsArray.algo;

	//void * workspace;
	void * workspace = NULL;
	hipMalloc(&workspace, workspaceBytes);


	// PREFORM MATMUL ON GPU

	float alpha = 1.0, beta = 0.0;
	struct timeval  tv1, tv2;
    gettimeofday(&tv1, NULL);

    hipDeviceSynchronize();

	status = hipblasLtMatmul(handle,
							matmulDesc,
							&alpha,
							d_A,
							Adesc,
							d_B,
							Bdesc,
							&beta,
							d_C,
							Cdesc,
							d_C,
							Ddesc,
							&algo,
							workspace,
							workspaceBytes,
							0);


	hipDeviceSynchronize();

	gettimeofday(&tv2, NULL);
    double time_taken = (double) (tv2.tv_usec - tv1.tv_usec) / 1000000 +
         					(double) (tv2.tv_sec - tv1.tv_sec); // in seconds

    printf("SGEMM where: m=%zu, k=%zu, n=%zu took --- %f seconds\n", M, K, N, time_taken);


    hipMemcpy(C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost);

    fpC = fopen(pathC, "w+");
	free(pathC);
	fwrite(C, sizeof(float), M * N, fpC);
	fclose(fpC);


    /* CLEAN UP MEMORY */

    // FREE workspace
	hipFree(workspace);

	// FREE cuBlasLt Structs 
	status = hipblasLtMatmulPreferenceDestroy(pref);
	status = hipblasLtMatmulDescDestroy(matmulDesc);

	status = hipblasLtMatrixLayoutDestroy(Adesc);
	status = hipblasLtMatrixLayoutDestroy(Bdesc);
	status = hipblasLtMatrixLayoutDestroy(Cdesc);
	status = hipblasLtMatrixLayoutDestroy(Ddesc);

	status = hipblasLtDestroy(handle);


	// FREE MATRICES
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	free(C);

	return 0;

}